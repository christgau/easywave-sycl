#include "hip/hip_runtime.h"
/*
 * EasyWave - A realtime tsunami simulation program with GPU support.
 * Copyright (C) 2014  Andrey Babeyko, Johannes Spazier
 * GFZ German Research Centre for Geosciences (http://www.gfz-potsdam.de)
 *
 * Parts of this program (especially the GPU extension) were developed
 * within the context of the following publicly funded project:
 * - TRIDEC, EU 7th Framework Programme, Grant Agreement 258723
 *   (http://www.tridec-online.eu)
 *
 * Licensed under the EUPL, Version 1.1 or - as soon they will be approved by
 * the European Commission - subsequent versions of the EUPL (the "Licence"),
 * complemented with the following provision: For the scientific transparency
 * and verification of results obtained and communicated to the public after
 * using a modified version of the work, You (as the recipient of the source
 * code and author of this modified version, used to produce the published
 * results in scientific communications) commit to make this modified source
 * code available in a repository that is easily and freely accessible for a
 * duration of five years after the communication of the obtained results.
 *
 * You may not use this work except in compliance with the Licence.
 *
 * You may obtain a copy of the Licence at:
 * https://joinup.ec.europa.eu/software/page/eupl
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the Licence is distributed on an "AS IS" basis,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the Licence for the specific language governing permissions and
 * limitations under the Licence.
 */

#include <string>

#include "hip/hip_runtime.h"
#include "ewGpuNode.hpp"
#include "ewKernels.cuda.cuh"

#define CUDA_CALL(x) if( (x) != hipSuccess ) { throw std::runtime_error("Error in file " __FILE__ ":" + std::to_string(__LINE__) + ": " + hipGetErrorString( hipGetLastError() ) ); }

CGpuNode::CGpuNode() {

	pitch = 0;
	copied = true;

	int device;
	struct hipDeviceProp_t props;

	CUDA_CALL(hipGetDevice(&device));
	CUDA_CALL(hipGetDeviceProperties(&props, device));

	std::cout << "Selected device: [" << device << "]: " << props.name << std::endl;

	have_profiling = true;
	for (auto &kd: kernel_duration) {
		kd = 0;
	}
}

CGpuNode::~CGpuNode()
{
	if (have_profiling) {
		dumpProfilingData();
	}
}

int CGpuNode::mallocMem() {

	CArrayNode::mallocMem();

	Params& dp = data.params;

	/* fill in some fields here */
	dp.nI = NLon;
	dp.nJ = NLat;
	dp.sshArrivalThreshold = Par.sshArrivalThreshold;
	dp.sshClipThreshold = Par.sshClipThreshold;
	dp.sshZeroThreshold = Par.sshZeroThreshold;
	dp.lpad = 31;

	size_t nJ_aligned = dp.nJ + dp.lpad;

	/* 2-dim */
	CUDA_CALL( hipMallocPitch( &(data.d), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.h), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.hMax), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.fM), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.fN), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.cR1), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.cR2), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.cR4), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	CUDA_CALL( hipMallocPitch( &(data.tArr), &pitch, nJ_aligned * sizeof(float), dp.nI ) );
	/* TODO: cR3, cR5 for coriolis */

	/* 1-dim */
	CUDA_CALL( hipMalloc( &(data.cR6), dp.nJ * sizeof(float) ) );
	CUDA_CALL( hipMalloc( &(data.cB1), dp.nI * sizeof(float) ) );
	CUDA_CALL( hipMalloc( &(data.cB2), dp.nJ * sizeof(float) ) );
	CUDA_CALL( hipMalloc( &(data.cB3), dp.nI * sizeof(float) ) );
	CUDA_CALL( hipMalloc( &(data.cB4), dp.nJ * sizeof(float) ) );

	CUDA_CALL( hipMalloc( &(data.g_MinMax), sizeof(int4) ) );

	/* TODO: make sure that pitch is a multiple of 4 and the same for each hipMallocPitch() call */
	dp.pI = pitch / sizeof(float);

	return 0;
}

int CGpuNode::copyToGPU() {

	Params& dp = data.params;

	/* align left grid boundary to a multiple of 32 with an offset 1 */
        Jmin -= (Jmin-2) % MEM_ALIGN;

        /* fill in further fields here */
        dp.iMin = Imin;
	dp.iMax = Imax;
        dp.jMin = Jmin;
	dp.jMax = Jmax;

	/* add offset to data.d to guarantee alignment: data.d + LPAD */
	/* 2-dim */
	CUDA_CALL( hipMemcpy2D( data.d + dp.lpad, pitch, d, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.h + dp.lpad, pitch, h, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.hMax + dp.lpad, pitch, hMax, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.fM + dp.lpad, pitch, fM, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.fN + dp.lpad, pitch, fN, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.cR1 + dp.lpad, pitch, cR1, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.cR2 + dp.lpad, pitch, cR2, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.cR4 + dp.lpad, pitch, cR4, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy2D( data.tArr + dp.lpad, pitch, tArr, dp.nJ * sizeof(float), dp.nJ * sizeof(float), dp.nI, hipMemcpyHostToDevice ) );

	/* FIXME: move global variables into data structure */
	/* 1-dim */
	CUDA_CALL( hipMemcpy( data.cR6, R6, dp.nJ * sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy( data.cB1, C1, dp.nI * sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy( data.cB2, C2, dp.nJ * sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy( data.cB3, C3, dp.nI * sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_CALL( hipMemcpy( data.cB4, C4, dp.nJ * sizeof(float), hipMemcpyHostToDevice ) );

	return 0;
}
int CGpuNode::copyFromGPU() {

	Params& dp = data.params;

	CUDA_CALL( hipMemcpy2D( hMax, dp.nJ * sizeof(float), data.hMax + dp.lpad, pitch, dp.nJ * sizeof(float), dp.nI, hipMemcpyDeviceToHost ) );
	CUDA_CALL( hipMemcpy2D( tArr, dp.nJ * sizeof(float), data.tArr + dp.lpad, pitch, dp.nJ * sizeof(float), dp.nI, hipMemcpyDeviceToHost ) );

	return 0;
}

int CGpuNode::copyIntermediate() {

	/* ignore copy requests if data already present on CPU side */
	if( copied )
		return 0;

	Params& dp = data.params;

	CUDA_CALL( hipMemcpy2D( h, dp.nJ * sizeof(float), data.h + dp.lpad, pitch, dp.nJ * sizeof(float), dp.nI, hipMemcpyDeviceToHost ) );

	/* copy finished */
	copied = true;

	return 0;
}

int CGpuNode::copyPOIs() {

	Params& dp = data.params;

	if( copied )
		return 0;

	for( int n = 0; n < NPOIs; n++ ) {

		int i = idxPOI[n] / dp.nJ + 1;
		int j = idxPOI[n] % dp.nJ + 1;

		int id = data.idx( i, j );

		CUDA_CALL( hipMemcpy( h + idxPOI[n], data.h + dp.lpad + id, sizeof(float), hipMemcpyDeviceToHost ) );
	}

	return 0;
}

int CGpuNode::freeMem() {

	/* 2-dim */
	CUDA_CALL( hipFree( data.d ) );
	CUDA_CALL( hipFree( data.h ) );
	CUDA_CALL( hipFree( data.hMax ) );
	CUDA_CALL( hipFree( data.fM ) );
	CUDA_CALL( hipFree( data.fN ) );
	CUDA_CALL( hipFree( data.cR1 ) );
	CUDA_CALL( hipFree( data.cR2 ) );
	CUDA_CALL( hipFree( data.cR4 ) );
	CUDA_CALL( hipFree( data.tArr ) );

	/* 1-dim */
	CUDA_CALL( hipFree( data.cR6 ) );
	CUDA_CALL( hipFree( data.cB1 ) );
	CUDA_CALL( hipFree( data.cB2 ) );
	CUDA_CALL( hipFree( data.cB3 ) );
	CUDA_CALL( hipFree( data.cB4 ) );

	CUDA_CALL( hipFree( data.g_MinMax ) );

	CArrayNode::freeMem();

	return 0;
}

int CGpuNode::run() {

	Params& dp = data.params;

	static hipEvent_t evtStart[NUM_TIMED_KERNELS];
	static hipEvent_t evtEnd[NUM_TIMED_KERNELS];
	static bool events_initialized = false;

	if (!events_initialized) {
	        for (int i = 0; i < NUM_TIMED_KERNELS; i++) {
        	        hipEventCreate(&(evtStart[i]));
                	hipEventCreate(&(evtEnd[i]));
	        }

		events_initialized = true;
	}

	int nThreads = 256;
	int xThreads = 32;
	int yThreads = nThreads / xThreads;

	int NJ = dp.jMax - dp.jMin + 1;
	int NI = dp.iMax - dp.iMin + 1;
	int xBlocks = ceil( (float)NJ / (float)xThreads );
	int yBlocks = ceil( (float)NI / (float)yThreads );

	dim3 threads( xThreads, yThreads );
	dim3 blocks( xBlocks, yBlocks );

	int nBlocks = ceil( (float)max(dp.nI,dp.nJ) / (float)nThreads );

	dp.mTime = Par.time;

	CUDA_CALL( hipEventRecord( evtStart[KERNEL_WAVE_UPDATE], 0 ) );
	waveUpdateKernel<<<blocks,threads>>>( data );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_WAVE_UPDATE], 0 ) );

	CUDA_CALL( hipEventRecord( evtStart[KERNEL_WAVE_BOUND], 0 ) );
	waveBoundaryKernel<<<nBlocks,nThreads>>>( data );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_WAVE_BOUND], 0 ) );

	CUDA_CALL( hipEventRecord( evtStart[KERNEL_FLUX_UPDATE], 0 ) );
	fluxUpdateKernel<<<blocks,threads>>>( data );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_FLUX_UPDATE], 0 ) );

	CUDA_CALL( hipEventRecord( evtStart[KERNEL_FLUX_BOUND], 0 ) );
	fluxBoundaryKernel<<<nBlocks,nThreads>>>( data );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_FLUX_BOUND], 0 ) );

	CUDA_CALL( hipEventRecord( evtStart[KERNEL_MEMSET], 0 ) );
	CUDA_CALL( hipMemset( data.g_MinMax, 0, sizeof(data.g_MinMax) ) );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_MEMSET], 0 ) );

	CUDA_CALL( hipEventRecord( evtStart[KERNEL_EXTEND], 0 ) );
	gridExtendKernel<<<nBlocks,nThreads>>>( data );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_EXTEND], 0 ) );

	int4 MinMax;
	CUDA_CALL( hipEventRecord( evtStart[KERNEL_MEMCPY], 0 ) );
	CUDA_CALL( hipMemcpy( &MinMax, data.g_MinMax, sizeof(MinMax), hipMemcpyDeviceToHost ) );
	CUDA_CALL( hipEventRecord( evtEnd[KERNEL_MEMCPY], 0 ) );
	hipDeviceSynchronize();

	if( MinMax.x ) Imin = dp.iMin = max( dp.iMin-1, 2 );
	if( MinMax.y ) Imax = dp.iMax = min( dp.iMax+1, dp.nI-1 );
	if( MinMax.z ) Jmin = dp.jMin = max( dp.jMin-MEM_ALIGN, 2 );
	if( MinMax.w ) Jmax = dp.jMax = min( dp.jMax+1, dp.nJ-1 );

	float duration;
	for( int j = 0; j < NUM_TIMED_KERNELS; j++ ) {
		hipEventElapsedTime( &duration, evtStart[j], evtEnd[j]);
		kernel_duration[j] += duration;
	}

	/* data has changed now -> copy becomes necessary */
	copied = false;

	return 0;
}
